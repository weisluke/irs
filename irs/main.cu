#include "hip/hip_runtime.h"
/*****************************************************************

Please provide credit to Luke Weisenbach should this code be used.
Email: weisluke@alum.mit.edu

*****************************************************************/


#include "complex.cuh"
#include "irs_microlensing.cuh"
#include "star.cuh"
#include "util.hpp"

#include <hiprand/hiprand_kernel.h>

#include <algorithm>
#include <chrono>
#include <cmath>
#include <fstream>
#include <iostream>
#include <limits>
#include <new>
#include <string>


using dtype = float;

/*constants to be used*/
const dtype PI = static_cast<dtype>(3.1415926535898);
constexpr int OPTS_SIZE = 2 * 18;
const std::string OPTS[OPTS_SIZE] =
{
	"-h", "--help",
	"-k", "--kappa_tot",
	"-s", "--shear",
	"-t", "--theta_e",
	"-ks", "--kappa_star",
	"-r", "--rectangular",
	"-a", "--approx",
	"-ss", "--safety_scale",
	"-sf", "--starfile",
	"-hl", "--half_length",
	"-px", "--pixels",
	"-nr", "--num_rays",
	"-rs", "--random_seed",
	"-wm", "--write_maps",
	"-wp", "--write_parities",
	"-wh", "--write_histograms",
	"-ot", "--outfile_type",
	"-o", "--outfile_prefix"
};


/*default input option values*/
dtype kappa_tot = static_cast<dtype>(0.3);
dtype shear = static_cast<dtype>(0.3);
dtype theta_e = static_cast<dtype>(1);
dtype kappa_star = static_cast<dtype>(0.27);
int rectangular = 1;
int approx = 1;
dtype safety_scale = static_cast<dtype>(1.37);
std::string starfile = "";
dtype half_length = static_cast<dtype>(5);
int num_pixels = 1000;
int num_rays = 100;
int random_seed = 0;
int write_maps = 1;
int write_parities = 0;
int write_histograms = 1;
std::string outfile_type = ".bin";
std::string outfile_prefix = "./";

/*default derived parameter values
number of stars, upper and lower mass cutoffs,
<m>, and <m^2>*/
int num_stars = 0;
dtype m_lower = static_cast<dtype>(1);
dtype m_upper = static_cast<dtype>(1);
dtype mean_mass = static_cast<dtype>(1);
dtype mean_squared_mass = static_cast<dtype>(1);



/************************************
Print the program usage help message

\param name -- name of the executable
************************************/
void display_usage(char* name)
{
	if (name)
	{
		std::cout << "Usage: " << name << " opt1 val1 opt2 val2 opt3 val3 ...\n";
	}
	else
	{
		std::cout << "Usage: programname opt1 val1 opt2 val2 opt3 val3 ...\n";
	}
	std::cout
		<< "                                                                               \n"
		<< "Options:\n"
		<< "  -h,--help               Show this help message.\n"
		<< "  -k,--kappa_tot          Specify the total convergence. Default value: " << kappa_tot << "\n"
		<< "  -s,--shear              Specify the external shear. Default value: " << shear << "\n"
		<< "  -t,--theta_e            Specify the size of the Einstein radius of a unit\n"
		<< "                          mass point lens in arbitrary units. Default value: " << theta_e << "\n"
		<< "  -ks,--kappa_star        Specify the convergence in point mass lenses.\n"
		<< "                          Default value: " << kappa_star << "\n"
		<< "  -r,--rectangular        Specify whether the star field should be\n"
		<< "                          rectangular (1) or circular (0). Default value: " << rectangular << "\n"
		<< "  -a,--approx             Specify whether terms for alpha_smooth should be\n"
		<< "                          approximated (1) or exact (0). Default value: " << approx << "\n"
		<< "  -ss,--safety_scale      Specify the multiplicative safety factor over the\n"
		<< "                          shooting region to be used when generating the star\n"
		<< "                          field. Default value: " << safety_scale << "\n"
		<< "  -sf,--starfile          Specify the location of a star positions and masses\n"
		<< "                          file. The file may be either a whitespace delimited\n"
		<< "                          text file containing valid double precision values\n"
		<< "                          for a star's x coordinate, y coordinate, and mass, in\n"
		<< "                          that order, on each line, or a binary file of star\n"
		<< "                          structures (as defined in this source code). If\n"
		<< "                          specified, the number of stars is determined through\n"
		<< "                          this file.\n"
		<< "  -hl,--half_length       Specify the half-length of the square source plane\n"
		<< "                          region to find the magnification in.\n"
		<< "                          Default value: " << half_length << "\n"
		<< "  -px,--pixels            Specify the number of pixels per side for the\n"
		<< "                          magnification map. Default value: " << num_pixels << "\n"
		<< "  -nr,--num_rays          Specify the average number of rays per pixel.\n"
		<< "                          Default value: " << num_rays << "\n"
		<< "  -rs,--random_seed       Specify the random seed for star field generation.\n"
		<< "                          A value of 0 is reserved for star input files.\n"
		<< "  -wm,--write_maps        Specify whether to write magnification maps (1) or\n"
		<< "                          not (0). Default value: " << write_maps << "\n"
		<< "  -wp,--write_parities    Specify whether to write parity specific\n"
		<< "                          magnification maps (1) or not (0). Default value: " << write_parities << "\n"
		<< "  -wh,--write_histograms  Specify whether to write histograms (1) or not (0).\n"
		<< "                          Default value: " << write_histograms << "\n"
		<< "  -ot,--outfile_type      Specify the type of file to be output. Valid options\n"
		<< "                          are binary (.bin) or text (.txt). Default value: " << outfile_type << "\n"
		<< "  -o,--outfile_prefix     Specify the prefix to be used in output file names.\n"
		<< "                          Default value: " << outfile_prefix << "\n"
		<< "                          Lines of .txt output files are whitespace delimited.\n"
		<< "                          Filenames are:\n"
		<< "                            irs_parameter_info     various parameter values\n"
		<< "                                                     used in calculations\n"
		<< "                            irs_stars              the first item is num_stars\n"
		<< "                                                     followed by binary\n"
		<< "                                                     representations of the\n"
		<< "                                                     star structures\n"
		<< "                            irs_numrays_numpixels  each line contains a number\n"
		<< "                                                     of rays and the number of\n"
		<< "                                                     pixels with that many rays\n"
		<< "                            irs_magnifications     the first item is num_pixels\n"
		<< "                                                     and the second item is\n"
		<< "                                                     num_pixels followed by the\n"
		<< "                                                     number of rays in each\n"
		<< "                                                     pixel\n";
}



int main(int argc, char* argv[])
{
	/*set precision for printing numbers to screen*/
	std::cout.precision(7);

	/*if help option has been input, display usage message*/
	if (cmd_option_exists(argv, argv + argc, "-h") || cmd_option_exists(argv, argv + argc, "--help"))
	{
		display_usage(argv[0]);
		return -1;
	}

	/*if there are input options, but not an even number (since all options
	take a parameter), display usage message and exit
	subtract 1 to take into account that first argument array value is program name*/
	if ((argc - 1) % 2 != 0)
	{
		std::cerr << "Error. Invalid input syntax.\n";
		display_usage(argv[0]);
		return -1;
	}

	/*check that all options given are valid. use step of 2 since all input
	options take parameters (assumed to be given immediately after the option)
	start at 1, since first array element, argv[0], is program name*/
	for (int i = 1; i < argc; i += 2)
	{
		if (!cmd_option_valid(OPTS, OPTS + OPTS_SIZE, argv[i]))
		{
			std::cerr << "Error. Invalid input syntax. Unknown option " << argv[i] << "\n";
			display_usage(argv[0]);
			return -1;
		}
	}


	/******************************************************************************
	BEGIN read in options and values, checking correctness and exiting if necessary
	******************************************************************************/

	char* cmdinput = nullptr;

	for (int i = 1; i < argc; i += 2)
	{
		cmdinput = cmd_option_value(argv, argv + argc, std::string(argv[i]));

		if (argv[i] == std::string("-k") || argv[i] == std::string("--kappa_tot"))
		{
			try
			{
				kappa_tot = static_cast<dtype>(std::stod(cmdinput));
			}
			catch (...)
			{
				std::cerr << "Error. Invalid kappa_tot input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-s") || argv[i] == std::string("--shear"))
		{
			try
			{
				shear = static_cast<dtype>(std::stod(cmdinput));;
			}
			catch (...)
			{
				std::cerr << "Error. Invalid shear input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-t") || argv[i] == std::string("--theta_e"))
		{
			try
			{
				theta_e = static_cast<dtype>(std::stod(cmdinput));;
				if (theta_e < std::numeric_limits<dtype>::min())
				{
					std::cerr << "Error. Invalid theta_e input. theta_e must be > " << std::numeric_limits<dtype>::min() << "\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid theta_e input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ks") || argv[i] == std::string("--kappa_star"))
		{
			try
			{
				kappa_star = static_cast<dtype>(std::stod(cmdinput));;
				if (kappa_star < std::numeric_limits<dtype>::min())
				{
					std::cerr << "Error. Invalid kappa_star input. kappa_star must be > " << std::numeric_limits<dtype>::min() << "\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid kappa_star input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-r") || argv[i] == std::string("--rectangular"))
		{
			try
			{
				rectangular = std::stoi(cmdinput);
				if (rectangular != 0 && rectangular != 1)
				{
					std::cerr << "Error. Invalid rectangular input. rectangular must be 1 (rectangular) or 0 (circular).\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid rectangular input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-a") || argv[i] == std::string("--approx"))
		{
			try
			{
				approx = std::stoi(cmdinput);
				if (approx != 0 && approx != 1)
				{
					std::cerr << "Error. Invalid approx input. approx must be 1 (approximate) or 0 (exact).\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid approx input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ss") || argv[i] == std::string("--safety_scale"))
		{
			try
			{
				safety_scale = static_cast<dtype>(std::stod(cmdinput));;
				if (safety_scale < 1)
				{
					std::cerr << "Error. Invalid safety_scale input. safety_scale must be > 1\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid safety_scale input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-sf") || argv[i] == std::string("--starfile"))
		{
			starfile = cmdinput;
		}
		else if (argv[i] == std::string("-hl") || argv[i] == std::string("--half_length"))
		{
			try
			{
				half_length = static_cast<dtype>(std::stod(cmdinput));;
				if (half_length < std::numeric_limits<dtype>::min())
				{
					std::cerr << "Error. Invalid half_length input. half_length must be > " << std::numeric_limits<dtype>::min() << "\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid half_length input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-px") || argv[i] == std::string("--pixels"))
		{
			try
			{
				num_pixels = std::stoi(cmdinput);
				if (num_pixels < 1)
				{
					std::cerr << "Error. Invalid num_pixels input. num_pixels must be an integer > 0\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid num_pixels input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-nr") || argv[i] == std::string("--num_rays"))
		{
			try
			{
				num_rays = std::stoi(cmdinput);
				if (num_rays < 1)
				{
					std::cerr << "Error. Invalid num_rays input. num_rays must be an integer > 0\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid num_rays input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-rs") || argv[i] == std::string("--random_seed"))
		{
			try
			{
				random_seed = std::stoi(cmdinput);
				if (random_seed == 0 && !(cmd_option_exists(argv, argv + argc, "-sf") || cmd_option_exists(argv, argv + argc, "--star_file")))
				{
					std::cerr << "Error. Invalid random_seed input. Seed of 0 is reserved for star input files.\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid random_seed input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-wm") || argv[i] == std::string("--write_maps"))
		{
			try
			{
				write_maps = std::stoi(cmdinput);
				if (write_maps != 0 && write_maps != 1)
				{
					std::cerr << "Error. Invalid write_maps input. write_maps must be 1 (true) or 0 (false).\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid write_maps input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-wp") || argv[i] == std::string("--write_parities"))
		{
			try
			{
				write_parities = std::stoi(cmdinput);
				if (write_parities != 0 && write_parities != 1)
				{
					std::cerr << "Error. Invalid write_parities input. write_parities must be 1 (true) or 0 (false).\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid write_parities input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-wh") || argv[i] == std::string("--write_histograms"))
		{
			try
			{
				write_histograms = std::stoi(cmdinput);
				if (write_histograms != 0 && write_histograms != 1)
				{
					std::cerr << "Error. Invalid write_histograms input. write_histograms must be 1 (true) or 0 (false).\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid write_histograms input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ot") || argv[i] == std::string("--outfile_type"))
		{
			outfile_type = cmdinput;
			if (outfile_type != ".bin" && outfile_type != ".txt")
			{
				std::cerr << "Error. Invalid outfile_type. outfile_type must be .bin or .txt\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-o") || argv[i] == std::string("--outfile_prefix"))
		{
			outfile_prefix = cmdinput;
		}
	}

	/****************************************************************************
	END read in options and values, checking correctness and exiting if necessary
	****************************************************************************/


	/*check that a CUDA capable device is present*/
	hipSetDevice(0);
	if (cuda_error("hipSetDevice", false, __FILE__, __LINE__)) return -1;


	/*if star file is specified, check validity of values and set num_stars, m_lower, m_upper,
	mean_mass, mean_squared_mass, and kappa_star based on star information*/
	if (starfile != "")
	{
		std::cout << "Calculating some parameter values based on star input file " << starfile << "\n";

		if (!read_star_params<dtype>(num_stars, m_lower, m_upper, mean_mass, mean_squared_mass, starfile))
		{
			std::cerr << "Error. Unable to read star field parameters from file " << starfile << "\n";
			return -1;
		}

		std::cout << "Done calculating some parameter values based on star input file " << starfile << "\n\n";
	}

	/*average magnification of the system*/
	dtype mu_ave = 1 / ((1 - kappa_tot) * (1 - kappa_tot) - shear * shear);

	/*number density of rays in the lens plane
	uses the fact that for a given user specified number density of rays
	in the source plane, further subdivisions are made that multiply the
	effective number of rays in the image plane by 27^2*/
	dtype num_rays_lens = num_rays / std::abs(mu_ave) * num_pixels * num_pixels / (2 * half_length * 2 * half_length) * 1 / (27 * 27);

	/*average separation between rays in one dimension is 1/sqrt(number density)*/
	dtype ray_sep = 1 / std::sqrt(num_rays_lens);

	/*shooting region is greater than outer boundary for macro-mapping by the
	size of the region of images visible for a macro-image which contain 99%
	of the flux*/
	dtype lens_hl_x1 = (half_length + 10 * theta_e * std::sqrt(kappa_star * mean_squared_mass / mean_mass)) / std::abs(1 - kappa_tot + shear);
	dtype lens_hl_x2 = (half_length + 10 * theta_e * std::sqrt(kappa_star * mean_squared_mass / mean_mass)) / std::abs(1 - kappa_tot - shear);

	/*make shooting region a multiple of the ray separation*/
	lens_hl_x1 = ray_sep * (static_cast<int>(lens_hl_x1 / ray_sep) + 1);
	lens_hl_x2 = ray_sep * (static_cast<int>(lens_hl_x2 / ray_sep) + 1);

	/*if stars are not drawn from external file, calculate final number of stars to use*/
	if (starfile == "")
	{
		if (rectangular)
		{
			num_stars = static_cast<int>((safety_scale * 2 * lens_hl_x1) * (safety_scale * 2 * lens_hl_x2) 
				* kappa_star / (PI * theta_e * theta_e * mean_mass)) + 1;
		}
		else
		{
			num_stars = static_cast<int>(safety_scale * safety_scale * (lens_hl_x1 * lens_hl_x1 + lens_hl_x2 * lens_hl_x2) 
				* kappa_star / (theta_e * theta_e * mean_mass)) + 1;
		}
	}

	std::cout << "Number of stars used: " << num_stars << "\n\n";

	Complex<dtype> c = std::sqrt(PI * theta_e * theta_e * num_stars * mean_mass / (4 * kappa_star))
		* Complex<dtype>(
			std::sqrt(std::abs((1 - kappa_tot - shear) / (1 - kappa_tot + shear))),
			std::sqrt(std::abs((1 - kappa_tot + shear) / (1 - kappa_tot - shear)))
			);
	dtype rad = std::sqrt(theta_e * theta_e * num_stars * mean_mass / kappa_star);

	int taylor = static_cast<int>(std::log(2 * kappa_star * c.abs() / (2 * half_length / num_pixels * PI)) / std::log(safety_scale));
	if (taylor < 1)
	{
		taylor = 1;
	}

	/**********************
	BEGIN memory allocation
	**********************/

	std::cout << "Beginning memory allocation...\n";

	hiprandState* states = nullptr;
	star<dtype>* stars = nullptr;
	int* pixels = nullptr;
	int* pixels_minima = nullptr;
	int* pixels_saddles = nullptr;

	/*allocate memory for stars*/
	hipMallocManaged(&states, num_stars * sizeof(hiprandState));
	if (cuda_error("hipMallocManaged(*states)", false, __FILE__, __LINE__)) return -1;
	hipMallocManaged(&stars, num_stars * sizeof(star<dtype>));
	if (cuda_error("hipMallocManaged(*stars)", false, __FILE__, __LINE__)) return -1;

	/*allocate memory for pixels*/
	hipMallocManaged(&pixels, num_pixels * num_pixels * sizeof(int));
	if (cuda_error("hipMallocManaged(*pixels)", false, __FILE__, __LINE__)) return -1;
	if (write_parities)
	{
		hipMallocManaged(&pixels_minima, num_pixels * num_pixels * sizeof(int));
		if (cuda_error("hipMallocManaged(*pixels_minima)", false, __FILE__, __LINE__)) return -1;
		hipMallocManaged(&pixels_saddles, num_pixels * num_pixels * sizeof(int));
		if (cuda_error("hipMallocManaged(*pixels_saddles)", false, __FILE__, __LINE__)) return -1;
	}

	std::cout << "Done allocating memory.\n\n";

	/********************
	END memory allocation
	********************/


	/*variables for kernel threads and blocks*/
	dim3 threads;
	dim3 blocks;

	/*number of threads per block, and number of blocks per grid
	uses 512 for number of threads in x dimension, as 1024 is the
	maximum allowable number of threads per block but is too large
	for some memory allocation, and 512 is next power of 2 smaller*/
	set_threads(threads, 512);
	set_blocks(threads, blocks, num_stars);


	/**************************
	BEGIN populating star array
	**************************/

	if (starfile == "")
	{
		std::cout << "Generating star field...\n";

		/*if random seed was not provided, get one based on the time*/
		if (random_seed == 0)
		{
			random_seed = static_cast<int>(std::chrono::system_clock::now().time_since_epoch().count());
		}

		/*generate random star field if no star file has been given
		uses default star mass of 1.0*/
		initialize_curand_states_kernel<dtype> <<<blocks, threads>>> (states, num_stars, random_seed);
		if (cuda_error("initialize_curand_states_kernel", true, __FILE__, __LINE__)) return -1;
		if (rectangular)
		{
			generate_rectangular_star_field_kernel<dtype> <<<blocks, threads>>> (states, stars, num_stars, c, static_cast<dtype>(1));
		}
		else
		{
			generate_circular_star_field_kernel<dtype> <<<blocks, threads>>> (states, stars, num_stars, rad, static_cast<dtype>(1));
		}
		if (cuda_error("generate_star_field_kernel", true, __FILE__, __LINE__)) return -1;

		std::cout << "Done generating star field.\n\n";
	}
	else
	{
		/*ensure random seed is 0 to denote that stars come from external file*/
		random_seed = 0;

		std::cout << "Reading star field from file " << starfile << "\n";

		/*reading star field from external file*/
		if (!read_star_file<dtype>(stars, num_stars, starfile))
		{
			std::cerr << "Error. Unable to read star field from file " << starfile << "\n";
			return -1;
		}

		std::cout << "Done reading star field from file " << starfile << "\n\n";
	}

	/************************
	END populating star array
	************************/


	/*redefine thread and block size to maximize parallelization*/
	set_threads(threads, 16, 16);
	set_blocks(threads, blocks, 2 * lens_hl_x1 / ray_sep, 2 * lens_hl_x2 / ray_sep);

	/*initialize pixel values*/
	initialize_pixels_kernel<dtype> <<<blocks, threads>>> (pixels, num_pixels);
	if (cuda_error("initialize_pixels_kernel", true, __FILE__, __LINE__)) return -1;
	if (write_parities)
	{
		initialize_pixels_kernel<dtype> <<<blocks, threads>>> (pixels_minima, num_pixels);
		if (cuda_error("initialize_pixels_kernel", true, __FILE__, __LINE__)) return -1;
		initialize_pixels_kernel<dtype> <<<blocks, threads>>> (pixels_saddles, num_pixels);
		if (cuda_error("initialize_pixels_kernel", true, __FILE__, __LINE__)) return -1;
	}


	/*start and end time for timing purposes*/
	std::chrono::high_resolution_clock::time_point starttime;
	std::chrono::high_resolution_clock::time_point endtime;

	std::cout << "Shooting rays...\n";
	/*get current time at start*/
	starttime = std::chrono::high_resolution_clock::now();
	shoot_rays_kernel<dtype> <<<blocks, threads>>> (kappa_tot, shear, theta_e, stars, num_stars, kappa_star, rectangular, c, approx, taylor, lens_hl_x1, lens_hl_x2, ray_sep, half_length, pixels_minima, pixels_saddles, pixels, num_pixels);
	if (cuda_error("shoot_rays_kernel", true, __FILE__, __LINE__)) return -1;
	/*get current time at end of loop, and calculate duration in milliseconds*/
	endtime = std::chrono::high_resolution_clock::now();
	double t_ray_shoot = std::chrono::duration_cast<std::chrono::milliseconds>(endtime - starttime).count() / 1000.0;
	std::cout << "Done shooting rays. Elapsed time: " << t_ray_shoot << " seconds.\n\n";


	/********************************
	create histograms of pixel values
	********************************/

	int* min_rays = nullptr;
	int* max_rays = nullptr;

	int* histogram = nullptr;
	int* histogram_minima = nullptr;
	int* histogram_saddles = nullptr;

	int histogram_length = 0;

	if (write_histograms)
	{
		std::cout << "Creating histograms...\n";

		hipMallocManaged(&min_rays, sizeof(int));
		if (cuda_error("hipMallocManaged(*min_rays)", false, __FILE__, __LINE__)) return -1;
		hipMallocManaged(&max_rays, sizeof(int));
		if (cuda_error("hipMallocManaged(*max_rays)", false, __FILE__, __LINE__)) return -1;

		*min_rays = std::numeric_limits<int>::max();
		*max_rays = 0;

		/*redefine thread and block size to maximize parallelization*/
		set_threads(threads, 16, 16);
		set_blocks(threads, blocks, num_pixels, num_pixels);

		histogram_min_max_kernel<dtype> <<<blocks, threads>>> (pixels, num_pixels, min_rays, max_rays);
		if (cuda_error("histogram_min_max_kernel", true, __FILE__, __LINE__)) return -1;
		if (write_parities)
		{
			histogram_min_max_kernel<dtype> <<<blocks, threads>>> (pixels_minima, num_pixels, min_rays, max_rays);
			if (cuda_error("histogram_min_max_kernel", true, __FILE__, __LINE__)) return -1;
			histogram_min_max_kernel<dtype> <<<blocks, threads>>> (pixels_saddles, num_pixels, min_rays, max_rays);
			if (cuda_error("histogram_min_max_kernel", true, __FILE__, __LINE__)) return -1;
		}

		histogram_length = *max_rays - *min_rays + 1;

		hipMallocManaged(&histogram, histogram_length * sizeof(int));
		if (cuda_error("hipMallocManaged(*histogram)", false, __FILE__, __LINE__)) return -1;
		if (write_parities)
		{
			hipMallocManaged(&histogram_minima, histogram_length * sizeof(int));
			if (cuda_error("hipMallocManaged(*histogram_minima)", false, __FILE__, __LINE__)) return -1;
			hipMallocManaged(&histogram_saddles, histogram_length * sizeof(int));
			if (cuda_error("hipMallocManaged(*histogram_saddles)", false, __FILE__, __LINE__)) return -1;
		}

		/*redefine thread and block size to maximize parallelization*/
		set_threads(threads, 512);
		set_blocks(threads, blocks, histogram_length);

		initialize_histogram_kernel<dtype> <<<blocks, threads>>> (histogram, histogram_length);
		if (cuda_error("initialize_histogram_kernel", true, __FILE__, __LINE__)) return -1;
		if (write_parities)
		{
			initialize_histogram_kernel<dtype> <<<blocks, threads>>> (histogram_minima, histogram_length);
			if (cuda_error("initialize_histogram_kernel", true, __FILE__, __LINE__)) return -1;
			initialize_histogram_kernel<dtype> <<<blocks, threads>>> (histogram_saddles, histogram_length);
			if (cuda_error("initialize_histogram_kernel", true, __FILE__, __LINE__)) return -1;
		}

		/*redefine thread and block size to maximize parallelization*/
		set_threads(threads, 16, 16);
		set_blocks(threads, blocks, num_pixels, num_pixels);

		histogram_kernel<dtype> <<<blocks, threads>>> (pixels, num_pixels, *min_rays, histogram);
		if (cuda_error("histogram_kernel", true, __FILE__, __LINE__)) return -1;
		if (write_parities)
		{
			histogram_kernel<dtype> <<<blocks, threads>>> (pixels_minima, num_pixels, *min_rays, histogram_minima);
			if (cuda_error("histogram_kernel", true, __FILE__, __LINE__)) return -1;
			histogram_kernel<dtype> <<<blocks, threads>>> (pixels_saddles, num_pixels, *min_rays, histogram_saddles);
			if (cuda_error("histogram_kernel", true, __FILE__, __LINE__)) return -1;
		}

		std::cout << "Done creating histograms.\n\n";
	}
	/***************************************
	done creating histograms of pixel values
	***************************************/


	/*stream for writing output files
	set precision to 9 digits*/
	std::ofstream outfile;
	outfile.precision(9);
	std::string fname;


	std::cout << "Writing parameter info...\n";
	fname = outfile_prefix + "irs_parameter_info.txt";
	outfile.open(fname);
	if (!outfile.is_open())
	{
		std::cerr << "Error. Failed to open file " << fname << "\n";
		return -1;
	}
	outfile << "kappa_tot " << kappa_tot << "\n";
	outfile << "shear " << shear << "\n";
	outfile << "mu_ave " << mu_ave << "\n";
	outfile << "theta_e " << theta_e << "\n";
	outfile << "kappa_star " << kappa_star << "\n";
	outfile << "lower_mass_cutoff " << m_lower << "\n";
	outfile << "upper_mass_cutoff " << m_upper << "\n";
	outfile << "mean_mass " << mean_mass << "\n";
	outfile << "mean_squared_mass " << mean_squared_mass << "\n";
	outfile << "num_stars " << num_stars << "\n";
	if (rectangular)
	{
		outfile << "corner_x1 " << c.re << "\n";
		outfile << "corner_x2 " << c.im << "\n";
		if (approx)
		{
			outfile << "taylor " << taylor << "\n";
		}
	}
	else
	{
		outfile << "rad " << rad << "\n";
	}
	outfile << "safety_scale " << safety_scale << "\n";
	outfile << "half_length " << half_length << "\n";
	outfile << "num_pixels " << num_pixels << "\n";
	outfile << "mean_rays_per_pixel " << num_rays << "\n";
	outfile << "random_seed " << random_seed << "\n";
	outfile << "lens_hl_x1 " << lens_hl_x1 << "\n";
	outfile << "lens_hl_x2 " << lens_hl_x2 << "\n";
	outfile << "ray_sep " << ray_sep << "\n";
	outfile << "t_ray_shoot " << t_ray_shoot << "\n";
	outfile.close();
	std::cout << "Done writing parameter info to file " << fname << "\n\n";


	std::cout << "Writing star info...\n";
	fname = outfile_prefix + "irs_stars" + outfile_type;
	if (!write_star_file<dtype>(stars, num_stars, fname))
	{
		std::cerr << "Error. Unable to write star info to file " << fname << "\n";
		return -1;
	}
	std::cout << "Done writing star info to file " << fname << "\n\n";


	/*histograms of magnification maps*/
	if (write_histograms)
	{
		std::cout << "Writing magnification histograms...\n";

		fname = outfile_prefix + "irs_numrays_numpixels.txt";
		if (!write_histogram<dtype>(histogram, histogram_length, *min_rays, fname))
		{
			std::cerr << "Error. Unable to write magnification histogram to file " << fname << "\n";
			return -1;
		}
		std::cout << "Done writing magnification histogram to file " << fname << "\n";
		if (write_parities)
		{
			fname = outfile_prefix + "irs_numrays_numpixels_minima.txt";
			if (!write_histogram<dtype>(histogram_minima, histogram_length, *min_rays, fname))
			{
				std::cerr << "Error. Unable to write magnification histogram to file " << fname << "\n";
				return -1;
			}
			std::cout << "Done writing magnification histogram to file " << fname << "\n";

			fname = outfile_prefix + "irs_numrays_numpixels_saddles.txt";
			if (!write_histogram<dtype>(histogram_saddles, histogram_length, *min_rays, fname))
			{
				std::cerr << "Error. Unable to write magnification histogram to file " << fname << "\n";
				return -1;
			}
			std::cout << "Done writing magnification histogram to file " << fname << "\n";
		}
		std::cout << "\n";
	}


	/*write magnifications for minima, saddle, and combined maps*/
	if (write_maps)
	{
		std::cout << "Writing magnifications...\n";

		fname = outfile_prefix + "irs_magnifications" + outfile_type;
		if (!write_array<int>(pixels, num_pixels, num_pixels, fname))
		{
			std::cerr << "Error. Unable to write magnifications to file " << fname << "\n";
			return -1;
		}
		std::cout << "Done writing magnifications to file " << fname << "\n";
		if (write_parities)
		{
			fname = outfile_prefix + "irs_magnifications_minima" + outfile_type;
			if (!write_array<int>(pixels_minima, num_pixels, num_pixels, fname))
			{
				std::cerr << "Error. Unable to write magnifications to file " << fname << "\n";
				return -1;
			}
			std::cout << "Done writing magnifications to file " << fname << "\n";

			fname = outfile_prefix + "irs_magnifications_saddles" + outfile_type;
			if (!write_array<int>(pixels_saddles, num_pixels, num_pixels, fname))
			{
				std::cerr << "Error. Unable to write magnifications to file " << fname << "\n";
				return -1;
			}
			std::cout << "Done writing magnifications to file " << fname << "\n";
		}
		std::cout << "\n";
	}

	std::cout << "Done.\n";

	hipDeviceReset();
	if (cuda_error("hipDeviceReset", false, __FILE__, __LINE__)) return -1;

	return 0;
}

