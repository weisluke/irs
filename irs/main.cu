#include "hip/hip_runtime.h"
/******************************************************************************

Please provide credit to Luke Weisenbach should this code be used.
Email: weisluke@alum.mit.edu

******************************************************************************/


#if defined(IRS_map) && !defined(IPM_map)
#include "irs.cuh"
#elif !defined(IRS_map) && defined(IPM_map)
#include "ipm.cuh"
#else
#error "Error. One, and only one, of IPM_map or IRS_map must be defined"
#endif
#include "util.cuh"

#include <iostream>
#include <string>


using dtype = float; //type to be used throughout this program. int, float, or double
#if defined(IRS_map) && !defined(IPM_map)
IRS<dtype> map_maker;
#elif !defined(IRS_map) && defined(IPM_map)
IPM<dtype> map_maker;
#else
#error "Error. One, and only one, of IPM_map or IRS_map must be defined"
#endif

/******************************************************************************
constants to be used
******************************************************************************/
constexpr int OPTS_SIZE = 2 * 26;
const std::string OPTS[OPTS_SIZE] =
{
	"-h", "--help",
	"-v", "--verbose",
	"-k", "--kappa_tot",
	"-y", "--shear",
	"-s", "--smooth_fraction", //provided as a courtesy in this executable. not part of the irs or ipm classes
	"-ks", "--kappa_star",
	"-t", "--theta_e",
	"-mf", "--mass_function",
	"-ms", "--m_solar",
	"-ml", "--m_lower",
	"-mh", "--m_upper",
	"-ll", "--light_loss",
	"-r", "--rectangular",
	"-a", "--approx",
	"-ss", "--safety_scale",
	"-sf", "--starfile",
	"-cy1", "--center_y1",
	"-cy2", "--center_y2",
	"-hl", "--half_length",
	"-px", "--pixels",
	"-nr", "--num_rays",
	"-rs", "--random_seed",
	"-wm", "--write_maps",
	"-wp", "--write_parities",
	"-wh", "--write_histograms",
	"-o", "--outfile_prefix"
};

/******************************************************************************
default input option values
******************************************************************************/
bool verbose = false;
dtype smooth_fraction = static_cast<dtype>(1 - map_maker.kappa_star / map_maker.kappa_tot);



/******************************************************************************
Print the program usage help message

\param name -- name of the executable
******************************************************************************/
void display_usage(char* name)
{
	if (name)
	{
		std::cout << "Usage: " << name << " opt1 val1 opt2 val2 opt3 val3 ...\n";
	}
	else
	{
		std::cout << "Usage: programname opt1 val1 opt2 val2 opt3 val3 ...\n";
	}
	std::cout
		<< "                                                                               \n"
		<< "Options:\n"
		<< "  -h,--help               Show this help message.\n"
		<< "  -v,--verbose            Toggle verbose output. Takes no option value.\n"
		<< "  -k,--kappa_tot          Specify the total convergence. Default value: " << map_maker.kappa_tot << "\n"
		<< "  -y,--shear              Specify the shear. Default value: " << map_maker.shear << "\n"
		<< "  -s,--smooth_fraction    Specify the fraction of convergence due to smoothly\n"
		<< "                          distributed mass. Default value: " << smooth_fraction << "\n"
		<< "  -ks,--kappa_star        Specify the convergence in point mass lenses. If\n"
		<< "                          provided, this overrides any supplied value for the\n"
		<< "                          smooth fraction. Default value: " << map_maker.kappa_star << "\n"
		<< "  -t,--theta_e            Specify the size of the Einstein radius of a unit\n"
		<< "                          mass point lens in arbitrary units. Default value: " << map_maker.theta_e << "\n"
		<< "  -mf,--mass_function     Specify the mass function to use for the point mass\n"
		<< "                          lenses. Options are: equal, uniform, Salpeter, and\n"
		<< "                          Kroupa. Default value: " << map_maker.mass_function_str << "\n"
		<< "  -ms,--m_solar           Specify the solar mass in arbitrary units.\n"
		<< "                          Default value: " << map_maker.m_solar << "\n"
		<< "  -ml,--m_lower           Specify the lower mass cutoff in solar mass units.\n"
		<< "                          Default value: " << map_maker.m_lower << "\n"
		<< "  -mh,--m_upper           Specify the upper mass cutoff in solar mass units.\n"
		<< "                          Default value: " << map_maker.m_upper << "\n"
		<< "  -ll,--light_loss        Allowed average fraction of light lost due to scatter\n"
		<< "                          by the microlenses in the large deflection limit.\n"
		<< "                          Default value: " << map_maker.light_loss << "\n"
		<< "  -r,--rectangular        Specify whether the star field should be\n"
		<< "                          rectangular (1) or circular (0). Default value: " << map_maker.rectangular << "\n"
		<< "  -a,--approx             Specify whether terms for alpha_smooth should be\n"
		<< "                          approximated (1) or exact (0). Default value: " << map_maker.approx << "\n"
		<< "  -ss,--safety_scale      Specify the multiplicative safety factor over the\n"
		<< "                          shooting region to be used when generating the star\n"
		<< "                          field. Default value: " << map_maker.safety_scale << "\n"
		<< "  -sf,--starfile          Specify the location of a binary file containing\n"
		<< "                          values for num_stars, rectangular, corner, theta_e,\n"
		<< "                          and the star positions and masses, in an order as\n"
		<< "                          defined in this source code.\n"
		<< "  -cy1, --center_y1       Specify the y1 position of the center of the\n"
		<< "                          magnification map. Default value: " << map_maker.center_y.re << "\n"
		<< "  -cy2, --center_y2       Specify the y2 position of the center of the\n"
		<< "                          magnification map. Default value: " << map_maker.center_y.im << "\n"
		<< "  -hl,--half_length       Specify the half-length of the square source plane\n"
		<< "                          region to find the magnification in.\n"
		<< "                          Default value: " << map_maker.half_length_source << "\n"
		<< "  -px,--pixels            Specify the number of pixels per side for the\n"
		<< "                          magnification map. Default value: " << map_maker.num_pixels << "\n"
		<< "  -nr,--num_rays          Specify the average number of rays per pixel in the\n"
		<< "                          absence of lensing (i.e. the number of rays per pixel\n"
		<< "                          equal to unit magnification). Default value: " << map_maker.num_rays_source << "\n"
		<< "  -rs,--random_seed       Specify the random seed for star field generation.\n"
		<< "                          A value of 0 is reserved for star input files.\n"
		<< "  -wm,--write_maps        Specify whether to write magnification maps (1) or\n"
		<< "                          not (0). Default value: " << map_maker.write_maps << "\n"
		<< "  -wp,--write_parities    Specify whether to write parity specific\n"
		<< "                          magnification maps (1) or not (0). Default value: " << map_maker.write_parities << "\n"
		<< "  -wh,--write_histograms  Specify whether to write histograms (1) or not (0).\n"
		<< "                          Default value: " << map_maker.write_histograms << "\n"
		<< "  -o,--outfile_prefix     Specify the prefix to be used in output file names.\n"
		<< "                          Default value: " << map_maker.outfile_prefix << "\n";
}



int main(int argc, char* argv[])
{
	/******************************************************************************
	if help option has been input, display usage message
	******************************************************************************/
	if (cmd_option_exists(argv, argv + argc, "-h") || cmd_option_exists(argv, argv + argc, "--help"))
	{
		display_usage(argv[0]);
		return -1;
	}

	/******************************************************************************
	if there are input options, but not an even number (since all options take a
	parameter), display usage message and exit
	subtract 1 to take into account that first argument array value is program name
	account for possible verbose option, which is a toggle and takes no input
	******************************************************************************/
	if ((argc - 1) % 2 != 0 &&
		!(cmd_option_exists(argv, argv + argc, "-v") || cmd_option_exists(argv, argv + argc, "--verbose")))
	{
		std::cerr << "Error. Invalid input syntax.\n";
		display_usage(argv[0]);
		return -1;
	}

	/******************************************************************************
	check that all options given are valid. use step of 2 since all input options
	take parameters (assumed to be given immediately after the option). start at 1,
	since first array element, argv[0], is program name
	account for possible verbose option, which is a toggle and takes no input
	******************************************************************************/
	for (int i = 1; i < argc; i += 2)
	{
		if (argv[i] == std::string("-v") || argv[i] == std::string("--verbose"))
		{
			verbose = true;
			i--;
			continue;
		}
		if (!cmd_option_valid(OPTS, OPTS + OPTS_SIZE, argv[i]))
		{
			std::cerr << "Error. Invalid input syntax. Unknown option " << argv[i] << "\n";
			display_usage(argv[0]);
			return -1;
		}
	}


	/******************************************************************************
	BEGIN read in options and values, checking correctness and exiting if necessary
	******************************************************************************/

	char* cmdinput = nullptr;

	for (int i = 1; i < argc; i += 2)
	{
		/******************************************************************************
		account for possible verbose option, which is a toggle and takes no input
		******************************************************************************/
		if (argv[i] == std::string("-v") || argv[i] == std::string("--verbose"))
		{
			i--;
			continue;
		}

		cmdinput = cmd_option_value(argv, argv + argc, std::string(argv[i]));

		if (argv[i] == std::string("-k") || argv[i] == std::string("--kappa_tot"))
		{
			try
			{
				set_param("kappa_tot", map_maker.kappa_tot, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid kappa_tot input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-y") || argv[i] == std::string("--shear"))
		{
			try
			{
				set_param("shear", map_maker.shear, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid shear input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-s") || argv[i] == std::string("--smooth_fraction"))
		{
			if (cmd_option_exists(argv, argv + argc, "-sf") || cmd_option_exists(argv, argv + argc, "--star_file"))
			{
				continue;
			}
			if (cmd_option_exists(argv, argv + argc, "-ks") || cmd_option_exists(argv, argv + argc, "--kappa_star"))
			{
				continue;
			}
			try
			{
				set_param("smooth_fraction", smooth_fraction, std::stod(cmdinput), verbose);
				if (smooth_fraction < 0)
				{
					std::cerr << "Error. Invalid smooth_fraction input. smooth_fraction must be >= 0\n";
					return -1;
				}
				else if (smooth_fraction >= 1)
				{
					std::cerr << "Error. Invalid smooth_fraction input. smooth_fraction must be < 1\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid smooth_fraction input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ks") || argv[i] == std::string("--kappa_star"))
		{
			if (cmd_option_exists(argv, argv + argc, "-sf") || cmd_option_exists(argv, argv + argc, "--star_file"))
			{
				continue;
			}
			try
			{
				set_param("kappa_star", map_maker.kappa_star, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid kappa_star input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-t") || argv[i] == std::string("--theta_e"))
		{
			if (cmd_option_exists(argv, argv + argc, "-sf") || cmd_option_exists(argv, argv + argc, "--star_file"))
			{
				continue;
			}
			try
			{
				set_param("theta_e", map_maker.theta_e, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid theta_e input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-mf") || argv[i] == std::string("--mass_function"))
		{
			if (cmd_option_exists(argv, argv + argc, "-sf") || cmd_option_exists(argv, argv + argc, "--star_file"))
			{
				continue;
			}
			set_param("mass_function", map_maker.mass_function_str, make_lowercase(cmdinput), verbose);
		}
		else if (argv[i] == std::string("-ms") || argv[i] == std::string("--m_solar"))
		{
			try
			{
				set_param("m_solar", map_maker.m_solar, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid m_solar input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ml") || argv[i] == std::string("--m_lower"))
		{
			if (cmd_option_exists(argv, argv + argc, "-sf") || cmd_option_exists(argv, argv + argc, "--star_file"))
			{
				continue;
			}
			try
			{
				set_param("m_lower", map_maker.m_lower, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid m_lower input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-mh") || argv[i] == std::string("--m_upper"))
		{
			if (cmd_option_exists(argv, argv + argc, "-sf") || cmd_option_exists(argv, argv + argc, "--star_file"))
			{
				continue;
			}
			try
			{
				set_param("m_upper", map_maker.m_upper, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid m_upper input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ll") || argv[i] == std::string("--light_loss"))
		{
			try
			{
				set_param("light_loss", map_maker.light_loss, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid light_loss input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-r") || argv[i] == std::string("--rectangular"))
		{
			if (cmd_option_exists(argv, argv + argc, "-sf") || cmd_option_exists(argv, argv + argc, "--star_file"))
			{
				continue;
			}
			try
			{
				set_param("rectangular", map_maker.rectangular, std::stoi(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid rectangular input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-a") || argv[i] == std::string("--approx"))
		{
			try
			{
				set_param("approx", map_maker.approx, std::stoi(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid approx input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ss") || argv[i] == std::string("--safety_scale"))
		{
			try
			{
				set_param("safety_scale", map_maker.safety_scale, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid safety_scale input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-sf") || argv[i] == std::string("--starfile"))
		{
			set_param("starfile", map_maker.starfile, cmdinput, verbose);
		}
		else if (argv[i] == std::string("-cy1") || argv[i] == std::string("--center_y1"))
		{
			try
			{
				set_param("center_y1", map_maker.center_y.re, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid center_y1 input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-cy2") || argv[i] == std::string("--center_y2"))
		{
			try
			{
				set_param("center_y2", map_maker.center_y.im, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid center_y2 input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-hl") || argv[i] == std::string("--half_length"))
		{
			try
			{
				set_param("half_length", map_maker.half_length_source, std::stod(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid half_length input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-px") || argv[i] == std::string("--pixels"))
		{
			try
			{
				set_param("num_pixels", map_maker.num_pixels, std::stoi(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid num_pixels input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-nr") || argv[i] == std::string("--num_rays"))
		{
			try
			{
				set_param("num_rays", map_maker.num_rays_source, std::stoi(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid num_rays input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-rs") || argv[i] == std::string("--random_seed"))
		{
			if (cmd_option_exists(argv, argv + argc, "-sf") || cmd_option_exists(argv, argv + argc, "--star_file"))
			{
				continue;
			}
			try
			{
				set_param("random_seed", map_maker.random_seed, std::stoi(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid random_seed input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-wm") || argv[i] == std::string("--write_maps"))
		{
			try
			{
				set_param("write_maps", map_maker.write_maps, std::stoi(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid write_maps input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-wp") || argv[i] == std::string("--write_parities"))
		{
			try
			{
				set_param("write_parities", map_maker.write_parities, std::stoi(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid write_parities input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-wh") || argv[i] == std::string("--write_histograms"))
		{
			try
			{
				set_param("write_histograms", map_maker.write_histograms, std::stoi(cmdinput), verbose);
			}
			catch (...)
			{
				std::cerr << "Error. Invalid write_histograms input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-o") || argv[i] == std::string("--outfile_prefix"))
		{
			set_param("outfile_prefix", map_maker.outfile_prefix, cmdinput, verbose);
		}
	}

	if (!(cmd_option_exists(argv, argv + argc, "-sf") || cmd_option_exists(argv, argv + argc, "--star_file")) &&
		!(cmd_option_exists(argv, argv + argc, "-ks") || cmd_option_exists(argv, argv + argc, "--kappa_star")))
	{
		set_param("kappa_star", map_maker.kappa_star, (1 - smooth_fraction) * map_maker.kappa_tot, verbose);
	}

	std::cout << "\n";

	/******************************************************************************
	END read in options and values, checking correctness and exiting if necessary
	******************************************************************************/


	/******************************************************************************
	check that a CUDA capable device is present
	******************************************************************************/
	int n_devices = 0;

	hipGetDeviceCount(&n_devices);
	if (cuda_error("hipGetDeviceCount", false, __FILE__, __LINE__)) return -1;

	if (verbose)
	{
		std::cout << "Available CUDA capable devices:\n\n";

		for (int i = 0; i < n_devices; i++)
		{
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, i);
			if (cuda_error("hipGetDeviceProperties", false, __FILE__, __LINE__)) return -1;

			show_device_info(i, prop);
		}
	}

	if (n_devices > 1)
	{
		std::cout << "More than one CUDA capable device detected. Defaulting to first device.\n\n";
	}
	hipSetDevice(0);
	if (cuda_error("hipSetDevice", false, __FILE__, __LINE__)) return -1;


	/******************************************************************************
	run and save files
	******************************************************************************/
	if (!map_maker.run(verbose)) return -1;
	if (!map_maker.save(verbose)) return -1;


	std::cout << "Done.\n";

	hipDeviceReset();
	if (cuda_error("hipDeviceReset", false, __FILE__, __LINE__)) return -1;

	return 0;
}

